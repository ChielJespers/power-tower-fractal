#include "hip/hip_runtime.h"
#include <gd.h>
#include <stdio.h>

#include <fenv.h>
#include <math.h>
#include <errno.h>

// ------------------------
// TO BE CUSTOMIZED BY USER
// ------------------------

// RENDERING PARAMETERS
#define sharpness     10000                                          // number of pixels specifying PNG pngWidth
#define maxIter       500                                          // set higher for highly zoomed-in pictures

// ------------------------
// COMPLEX DOMAIN
double reStart = -0.193;
double reEnd = -0.183;
double imStart = 0.23;
double imEnd = 0.24;

// See the bottom of this code for a discussion of some output possibilities.
char*   filename =   "ZoomIslesharp.png";

int** make2DintArray(int arraySizeX, int arraySizeY);
void free2DintArray(int** myArray, int arraySizeX);

__device__
double getArg(double im);

__global__
void fillColor(int n, int H, int W, int* color, int* grey, int blue) {
  // COMPLEX DOMAIN
  double reStart = -0.193;
  double reEnd = -0.183;
  double imStart = 0.23;
  double imEnd = 0.24;

  int T = blockIdx.x*blockDim.x + threadIdx.x;
  if (T >= n) return;

  int x = T % H;
  int y = T / H;
  double re = reStart + ((double) x / W * (reEnd - reStart));
  double im = imEnd - ((double) y / H * (imEnd - imStart));

  double nextRe, nextIm, logRe, logIm, powerRe, powerIm;

  int toggleOverflow = 0;                                          
  int numberOfIterations = 0;                                      
  if (re == 0 && im == 0){
    color[T] = blue;
  }
  else {
    logRe = .5*log(re*re + im*im);
    logIm = atan2(im, re);
    nextRe = re;
    nextIm = im;
    while (numberOfIterations < maxIter && toggleOverflow == 0)
    {
        powerRe = (nextRe * logRe - nextIm * logIm);
        powerIm = (nextRe * logIm + nextIm * logRe);

        if (powerRe > 700) {
            toggleOverflow = 1;
        }

        nextRe = exp(powerRe) * cos(powerIm);
        nextIm = exp(powerRe) * sin(powerIm);
        
        numberOfIterations += 1;
    }
  }

  int shade = 255 - ((numberOfIterations * 255) / maxIter);
  color[T] = grey[shade];
}

int main(){

  FILE*       outfile;                               // defined in stdio
  gdImagePtr  image;                                 // a GD image object
  int         i, T, x, y;                            // array subscripts
  int         blue, grey[256];       // red, all possible shades of grey
  int*        d_grey;

  int pngWidth = sharpness;
  int pngHeight = pngWidth * (imEnd - imStart) / (reEnd - reStart);
  int N = pngWidth * pngHeight;

  //int** color = make2DintArray(pngWidth, pngHeight);
  int* color = (int*) malloc(N*sizeof(int));
  int* d_color;

  printf("width: %i\n", pngWidth);
  printf("height: %i\n", pngHeight);

  image = gdImageCreate(pngWidth, pngHeight);

  blue  = gdImageColorAllocate(image, 0, 0, 255);
  
  for (i=0; i<256; i++){
    grey[i] = gdImageColorAllocate(image, i,i,i);
  }

  //void fillColor(int n, int H, int W, int* color, int* grey, int blue) {
  hipMalloc(&d_grey, 256*sizeof(int)); 
  hipMalloc(&d_color, N*sizeof(int));

  hipMemcpy(d_grey, grey, 256*sizeof(int), hipMemcpyHostToDevice);

  // Calculate power tower convergence / divergence
  fillColor<<<(pngWidth*pngHeight+255)/256, 256>>>(N, pngHeight, pngWidth, d_color, d_grey, blue);
  hipMemcpy(color, d_color, N*sizeof(int), hipMemcpyDeviceToHost);

  for (T=0; T<pngWidth*pngHeight; T++) {
    x = T % pngHeight;
    y = T / pngHeight;
    gdImageSetPixel(image, x, y, color[T]);
  }

  // Free 2D array
  free(color);
  hipFree(d_color);
  hipFree(d_grey);
  // Finally, write the image out to a file.
  printf("Creating output file '%s'.\n", filename);
  outfile = fopen(filename, "wb");
  gdImagePng(image, outfile);
  fclose(outfile);
}

int** make2DintArray(int arraySizeX, int arraySizeY) {  
  int** theArray;
  theArray = (int**) malloc(arraySizeX*sizeof(int*));
  for (int i = 0; i < arraySizeX; i++) {
    theArray[i] = (int*) malloc(arraySizeY*sizeof(int));
  }
  return theArray;  
}

void free2DintArray(int** myArray, int arraySizeX) {
  for (int i = 0; i < arraySizeX; i++){  
    free(myArray[i]);  
  }  
  free(myArray);    
}

__device__
double getArg(double im) {
  while (im > M_PI) {
      im -= 2*M_PI;
  }
  while (im <= -M_PI ) {
      im += 2*M_PI;
  }
  return im;
}